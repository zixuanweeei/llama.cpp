#include "hip/hip_runtime.h"
#include "common.cuh"
#include "softmax.cuh"

template <typename T>
static __device__ __forceinline__ float t2f32(T val) {
    return (float) val;
}

template <>
__device__ float __forceinline__ t2f32<half>(half val) {
    return __half2float(val);
}

template <bool vals_smem, int ncols_template, int block_size_template, typename T>
static __global__ void soft_max_f32(const float * x, const T * mask, float * dst, const int ncols_par, const int nrows_y, const float scale, const float max_bias, const float m0, const float m1, uint32_t n_head_log2) {
    const int ncols = ncols_template == 0 ? ncols_par : ncols_template;

    const int tid  = threadIdx.x;
    const int rowx = blockIdx.x;
    const int rowy = rowx % nrows_y; // broadcast the mask in the row dimension

    const int block_size = block_size_template == 0 ? blockDim.x : block_size_template;

    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;

    const float slope = get_alibi_slope(max_bias, rowx/nrows_y, n_head_log2, m0, m1);

    extern __shared__ float data_soft_max_f32[];
    float * buf_max = data_soft_max_f32; // shared memory buffer for inter-warp communication (for max values)
    float * buf_den = data_soft_max_f32 + WARP_SIZE; // shared memory buffer for inter-warp communication (for sum(exp(v - max)))
    // shared memory buffer to cache values between iterations:
    float * vals = vals_smem ? buf_max + 2 * WARP_SIZE : dst + (int64_t)rowx*ncols;

    float max_val = -INFINITY;
    float den = 1;

#pragma unroll
    for (int col0 = 0; col0 < ncols; col0 += block_size) {
        const int col = col0 + tid;

        if (ncols_template == 0 && col >= ncols) {
            break;
        }

        const int64_t ix = (int64_t)rowx*ncols + col;
        const int64_t iy = (int64_t)rowy*ncols + col;

        const float val = x[ix]*scale + (mask ? slope*t2f32(mask[iy]) : 0.0f);

        if (vals_smem) {
            vals[col] = val;
        }

        float last_max_val = max_val;
        max_val = max(max_val, val);
        den = (isinf(last_max_val) ? 0.F : den * expf(last_max_val - max_val)) +
              expf(val - max_val);
    }

#pragma unroll
    for (int radix = 1; radix < 32; radix <<= 1) {
        constexpr auto shfl_mask = static_cast<uint32_t>(-1);
        float butterflied_v = __shfl_xor_sync(shfl_mask, max_val, radix);
        float butterflied_d = __shfl_xor_sync(shfl_mask, den, radix);

        float this_max_val = max(max_val, butterflied_v);

        den = isinf(this_max_val) ? 0
                           : (den * exp(max_val - this_max_val) +
                              butterflied_d * exp(butterflied_v - this_max_val));
        max_val = this_max_val;
    }

    if (block_size > WARP_SIZE) {
        if (warp_id == 0) {
            buf_max[lane_id] = -INFINITY;
            buf_den[lane_id] = 1;
        }
        __syncthreads();

        if (lane_id == 0) {
            buf_max[warp_id] = max_val;
            buf_den[warp_id] = den;
        }
        __syncthreads();

        max_val = buf_max[lane_id];
        den = buf_den[lane_id];
        
#pragma unroll
        for (int radix = 1; radix < 32; radix <<= 1) {
            constexpr auto shfl_mask = static_cast<uint32_t>(-1);
            float butterflied_v = __shfl_xor_sync(shfl_mask, max_val, radix);
            float butterflied_d = __shfl_xor_sync(shfl_mask, den, radix);

            float this_max_val = max(max_val, butterflied_v);

            den = isinf(this_max_val) ? 0
                               : (den * exp(max_val - this_max_val) +
                                  butterflied_d * exp(butterflied_v - this_max_val));
            max_val = this_max_val;
        }
    }

    const float inv_sum = 1.0f / den;

#pragma unroll
    for (int col0 = 0; col0 < ncols; col0 += block_size) {
        const int col = col0 + tid;

        if (ncols_template == 0 && col >= ncols) {
            return;
        }

        const int64_t idst = (int64_t)rowx * ncols + col;
        const int64_t ix = (int64_t)rowx * ncols + col;
        const int64_t iy = (int64_t)rowy * ncols + col;

        const float val =
            vals_smem
                ? vals[col]
                : (x[ix] * scale + (mask ? slope * t2f32(mask[iy]) : 0.0f));

        dst[idst] = expf(val - max_val) * inv_sum;
    }
}

template<typename T>
static void soft_max_f32_cuda(const float * x, const T * mask, float * dst, const int ncols_x, const int nrows_x, const int nrows_y, const float scale, const float max_bias, hipStream_t stream) {
    int nth = WARP_SIZE;
    while (nth < ncols_x && nth < CUDA_SOFT_MAX_BLOCK_SIZE) nth *= 2;
    const dim3 block_dims(nth,     1, 1);
    const dim3 block_nums(nrows_x, 1, 1);
    const size_t shmem = (GGML_PAD(ncols_x, WARP_SIZE) + 2 * WARP_SIZE) * sizeof(float);
    static_assert(CUDA_SOFT_MAX_BLOCK_SIZE == 1024, "These values need to be adjusted.");

    const uint32_t n_head      = nrows_x/nrows_y;
    const uint32_t n_head_log2 = 1u << (uint32_t) floorf(log2f((float) n_head));

    const float m0 = powf(2.0f, -(max_bias       ) / n_head_log2);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_head_log2);

    // FIXME: this limit could be raised by ~2-4x on Ampere or newer
    if (shmem < ggml_cuda_info().devices[ggml_cuda_get_device()].smpb) {
        switch (ncols_x) {
            case 32:
                soft_max_f32<true, 32, 32><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 64:
                soft_max_f32<true, 64, 64><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 128:
                soft_max_f32<true, 128, 128><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 256:
                soft_max_f32<true, 256, 256><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 512:
                soft_max_f32<true, 512, 512><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 1024:
                soft_max_f32<true, 1024, 1024><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 2048:
                soft_max_f32<true, 2048, 1024><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            case 4096:
                soft_max_f32<true, 4096, 1024><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
            default:
                soft_max_f32<true, 0, 0><<<block_nums, block_dims, shmem, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
                break;
        }
    } else {
        const size_t shmem_low = 2 * WARP_SIZE * sizeof(float);
        soft_max_f32<false, 0, 0><<<block_nums, block_dims, shmem_low, stream>>>(x, mask, dst, ncols_x, nrows_y, scale, max_bias, m0, m1, n_head_log2);
    }
}

void ggml_cuda_op_soft_max(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    const float * src0_d = (const float *)src0->data;
    const void  * src1_d = src1 ? (const void *)src1->data : nullptr;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    GGML_ASSERT(!src1 || src1->type == GGML_TYPE_F16 || src1->type == GGML_TYPE_F32); // src1 contains mask and it is optional

    const int64_t ne00    = src0->ne[0];
    const int64_t nrows_x = ggml_nrows(src0);
    const int64_t nrows_y = src0->ne[1];

    float scale    = 1.0f;
    float max_bias = 0.0f;

    memcpy(&scale,    (float *) dst->op_params + 0, sizeof(float));
    memcpy(&max_bias, (float *) dst->op_params + 1, sizeof(float));

    const bool use_f16 = (src1 && src1->type == GGML_TYPE_F16);

    if (use_f16) {
        const half * src1_dd = (const half *)src1_d;

        soft_max_f32_cuda(src0_d, src1_dd, dst_d, ne00, nrows_x, nrows_y, scale, max_bias, stream);
    } else {
        const float * src1_dd = (const float *)src1_d;

        soft_max_f32_cuda(src0_d, src1_dd, dst_d, ne00, nrows_x, nrows_y, scale, max_bias, stream);
    }
}
